#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include "pfordelta/opt_p4.h"
#include "dictionary/Dictionary.h"
#include "buffer/FixedIntCounter.h"
#include "buffer/FixedLongCounter.h"
#include "util/ParseCommandLine.h"
#include "PostingsPool.h"
#include "Pointers.h"
#include "Config.h"
#include "InvertedIndex.h"
#include "intersection/SvS.h"
#include "intersection/WAND.h"


#ifndef RETRIEVAL_ALGO_ENUM_GUARD
#define RETRIEVAL_ALGO_ENUM_GUARD
typedef enum Algorithm Algorithm;
enum Algorithm {
  SVS = 0,
  WAND = 1
};
#endif

#define THREADS_PER_BLOCK 512 
#define THREADS_PER_BLOCK_GLOBALPAIRS 64
#define LINEARBLOCK 100

int main (int argc, char** args) {
  // Index path
  char* inputPath = getValueCL(argc, args, "-index");
  // Query path
  char* queryPath = getValueCL(argc, args, "-query");
  // Output path (optional)
  char* outputPath = getValueCL(argc, args, "-output");
  // Hits
  int hits = 1000;
  if(isPresentCL(argc, args, "-hits")) {
    hits = atoi(getValueCL(argc, args, "-hits"));
  }
  // Algorithm
  char* intersectionAlgorithm = getValueCL(argc, args, "-algorithm");
  Algorithm algorithm = SVS;

  // Algorithm is limited to the following list (case sensitive):
  // - SvS (conjunctive)
  // - WAND (disjunctive)
  if(!strcmp(intersectionAlgorithm, "SvS")) {
    algorithm = SVS;
  } else if(!strcmp(intersectionAlgorithm, "WAND")) {
    algorithm = WAND;
  } else {
    printf("Invalid algorithm (Options: SvS | WAND)\n");
    return;
  }

  // Read the inverted index
  InvertedIndex* index = readInvertedIndex(inputPath);

  // Read queries. Query file must be in the following format:
  // - First line: <number of queries: integer>
  // - <query id: integer> <query length: integer> <query text: string>
  // Note that, if a query term does not have a corresponding postings list,
  // then we drop the query term from the query. Empty queries are not evaluated.
  FixedIntCounter* queryLength = createFixedIntCounter(32768, 0);
  FixedIntCounter* idToIndexMap = createFixedIntCounter(32768, 0);
  FILE* fp = fopen(queryPath, "r");
  int totalQueries = 0, id, qlen, fqlen, j, pos, termid, i;
  char query[1024];
  fscanf(fp, "%d", &totalQueries);
  //unsigned int** queries = (unsigned int**) malloc(totalQueries * sizeof(unsigned int*));
  unsigned int* linearQ = (unsigned int*) malloc(100 * totalQueries * sizeof(unsigned int));
  int* linearQ_count = (unsigned int*) malloc(totalQueries * sizeof(unsigned int));

  int totalLen = 0;
  for(i = 0; i < totalQueries; i++) {
    fscanf(fp, "%d %d", &id, &qlen);
    //queries[i] = (unsigned int*) malloc(qlen * sizeof(unsigned int));
    pos = 0;
    fqlen = qlen;
    for(j = 0; j < qlen; j++) {
      fscanf(fp, "%s", query);
      termid = getTermId(index->dictionary, query);
      if(termid >= 0) {
        if(getStartPointer(index->pointers, termid) != UNDEFINED_POINTER) {
			linearQ[totalLen] = termid;
			totalLen++;
          //queries[i][pos++] = termid;
        } else {
          fqlen--;
        }
      } else {
        fqlen--;
      }
    }
    setFixedIntCounter(idToIndexMap, id, i);
    setFixedIntCounter(queryLength, id, fqlen);
	linearQ_count[i] = totalLen;
  }
  fclose(fp);

  if(outputPath) {
    fp = fopen(outputPath, "w");
  }

  // Evaluate queries by iterating over the queries that are not empty
  id = -1;

/////////////////////// CUDA Entry
  SvS_GPU_Entry(
  	queryLength, 
  	idToIndexMap, 
  	outputPath, 
  	index, 
  	fp, 
  	totalQueries,
  	linearQ,
  	linearQ_count,
  	totalLen);
//////////////////////

  if(outputPath) {
    fclose(fp);
  }
  for(i = 0; i < totalQueries; i++) {
    if(queries[i]) {
      free(queries[i]);
    }
  }
  free(queries);
  destroyFixedIntCounter(queryLength);
  destroyFixedIntCounter(idToIndexMap);
  destroyInvertedIndex(index);
  return 0;
}

__device__ int decompressDocidBlock_GPU(int* pool, unsigned int* outBlock, long pointer) {
  int pSegment = DECODE_SEGMENT(pointer);
  unsigned int pOffset = DECODE_OFFSET(pointer);

  unsigned int aux[BLOCK_SIZE*4];
  unsigned int* block = &pool[pOffset + 5];
  detailed_p4_decode(outBlock, block, aux, 1);

  return pool[pOffset + 3];
}

__device__ long nextPointer_GPU(int* pool, long pointer) {
  if(pointer == UNDEFINED_POINTER) {
    return UNDEFINED_POINTER;
  }
  int pSegment = DECODE_SEGMENT(pointer);
  unsigned int pOffset = DECODE_OFFSET(pointer);

  if(pool[pOffset + 1] == UNKNOWN_SEGMENT) {
    return UNDEFINED_POINTER;
  }

  return ENCODE_POINTER(pool[pOffset + 1],
                        pool[pOffset + 2]);
}

__device__ int* intersectPostingsLists_SvS_GPU(int* pool, long a, long b, int minDf) {
  int* set = (int*) calloc(minDf, sizeof(int));
  unsigned int* dataA = (unsigned int*) calloc(BLOCK_SIZE * 2, sizeof(unsigned int));
  unsigned int* dataB = (unsigned int*) calloc(BLOCK_SIZE * 2, sizeof(unsigned int));

  int cA = decompressDocidBlock_GPU(pool, dataA, a);
  int cB = decompressDocidBlock_GPU(pool, dataB, b);
  int iSet = 0, iA = 0, iB = 0;

  while(a != UNDEFINED_POINTER && b != UNDEFINED_POINTER) {
    if(dataB[iB] == dataA[iA]) {
      set[iSet++] = dataA[iA];
      iA++;
      iB++;
    }

    if(iA == cA) {
      a = nextPointer_GPU(pool, a);
      if(a == UNDEFINED_POINTER) {
        break;
      }
      memset(dataA, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
      cA = decompressDocidBlock_GPU(pool, dataA, a);
      iA = 0;
    }
    if(iB == cB) {
      b = nextPointer_GPU(pool, b);
      if(b == UNDEFINED_POINTER) {
        break;
      }
      memset(dataB, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
      cB = decompressDocidBlock_GPU(pool, dataB, b);
      iB = 0;
    }

    if(dataA[iA] < dataB[iB]) {
      if(dataA[cA - 1] < dataB[iB]) {
        iA = cA - 1;
      }
      while(dataA[iA] < dataB[iB]) {
        iA++;
        if(iA == cA) {
          a = nextPointer_GPU(pool, a);
          if(a == UNDEFINED_POINTER) {
            break;
          }
          memset(dataA, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
          cA = decompressDocidBlock_GPU(pool, dataA, a);
          iA = 0;
        }
        if(dataA[cA - 1] < dataB[iB]) {
          iA = cA - 1;
        }
      }
    } else {
      if(dataB[cB - 1] < dataA[iA]) {
        iB = cB - 1;
      }
      while(dataB[iB] < dataA[iA]) {
        iB++;
        if(iB == cB) {
          b = nextPointer_GPU(pool, b);
          if(b == UNDEFINED_POINTER) {
            break;
          }
          memset(dataB, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
          cB = decompressDocidBlock_GPU(pool, dataB, b);
          iB = 0;
        }
        if(dataB[cB - 1] < dataA[iA]) {
          iB = cB - 1;
        }
      }
    }
  }

  if(iSet < minDf) {
    set[iSet] = TERMINAL_DOCID;
  }

  free(dataA);
  free(dataB);

  return set;
}

__device__ int intersectSetPostingsList_SvS_GPU(int* pool, long a, int* currentSet, int len) {
  unsigned int* data = (unsigned int*) calloc(BLOCK_SIZE * 2, sizeof(unsigned int));
  int c = decompressDocidBlock_GPU(pool, data, a);
  int iSet = 0, iCurrent = 0, i = 0;

  while(a != UNDEFINED_POINTER && iCurrent < len) {
    if(currentSet[iCurrent] == TERMINAL_DOCID) {
      break;
    }
    if(data[i] == currentSet[iCurrent]) {
      currentSet[iSet++] = currentSet[iCurrent];
      iCurrent++;
      i++;
    }

    if(i == c) {
      a = nextPointer_GPU(pool, a);
      if(a == UNDEFINED_POINTER) {
        break;
      }
      memset(data, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
      c = decompressDocidBlock_GPU(pool, data, a);
      i = 0;
    }
    if(iCurrent == len) {
      break;
    }
    if(currentSet[iCurrent] == TERMINAL_DOCID) {
      break;
    }

    if(data[i] < currentSet[iCurrent]) {
      if(data[c - 1] < currentSet[iCurrent]) {
        i = c - 1;
      }
      while(data[i] < currentSet[iCurrent]) {
        i++;
        if(i == c) {
          a = nextPointer_GPU(pool, a);
          if(a == UNDEFINED_POINTER) {
            break;
          }
          memset(data, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
          c = decompressDocidBlock_GPU(pool, data, a);
          i = 0;
        }
        if(data[c - 1] < currentSet[iCurrent]) {
          i = c - 1;
        }
      }
    } else {
      while(currentSet[iCurrent] < data[i]) {
        iCurrent++;
        if(iCurrent == len) {
          break;
        }
        if(currentSet[iCurrent] == TERMINAL_DOCID) {
          break;
        }
      }
    }
  }

  if(iSet < len) {
    currentSet[iSet] = TERMINAL_DOCID;
  }

  free(data);
  return iSet;
}

__device__ int* intersectSvS_GPU(int* pool, long* startPointers, int len, int minDf) {
  if(len < 2) {
    unsigned int* block = (unsigned int*) calloc(BLOCK_SIZE * 2, sizeof(unsigned int));
    int* set = (int*) calloc(minDf, sizeof(int));
    int iSet = 0;
    long t = startPointers[0];
    while(t != UNDEFINED_POINTER) {
      memset(block, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
      int c = decompressDocidBlock_GPU(pool, block, t);
      memcpy(&set[iSet], block, c * sizeof(int));
      iSet += c;
      t = nextPointer_GPU(pool, t);
    }
    free(block);
    return set;
  }

  int* set = intersectPostingsLists_SvS_GPU(pool, startPointers[0], startPointers[1], minDf);
  int i;
  for(i = 2; i < len; i++) {
    intersectSetPostingsList_SvS_GPU(pool, startPointers[i], set, minDf);
  }
  return set;
}

__global__ void SvS_GPU(
	int* queryLength_counter,
	unsigned int queryLength_vocabSize,
	DefaultValue queryLength_defaultValue,	
	int* idToIndexMap_counter,
	unsigned int idToIndexMap_vocabSize,
	DefaultValue idToIndexMap_defaultValue,		
	int* index_df_counter,
	unsigned int index_df_vocabSize,
	DefaultValue index_df_defaultValue,		
	long* index_pointer_counter,
	unsigned int index_pointer_vocabSize,
	DefaultValue index_pointer_defaultValue,	
	int* index_pool_firstseg, //index->pool->pool[0]
	unsigned int index_pool_offset,
	unsigned int index_pool_segment,	
	unsigned int* linearQ,
	int* linearQ_count,
	int totalQuery){

	int id = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
	if(id >= queryLength_vocabSize) {
	  return;
	}
	
	if(queryLength_counter[id] == queryLength_defaultValue) {
	  return;
	}

	  //printf("id = %d\n", id);
	  // Measure elapsed time
	  int i, j;
	  int qlen = queryLength_counter[id];
	  int qindex = idToIndexMap_counter[id];
	  if (qindex > totalQuery){
	  	printf("Exceed the range!\n");
		return;
	  }
	  
	  unsigned int* qdf = (unsigned int*) calloc(qlen, sizeof(unsigned int));
	  int* sortedDfIndex = (int*) calloc(qlen, sizeof(int));
	  long* qStartPointers = (long*) calloc(qlen, sizeof(long));
	  int end = linearQ_count[qindex];
	  int start = 0;
	  if (qindex > 0){
		start = linearQ_count[qindex-1];
	  }
	  
	  if (linearQ[start]>= index_df_vocabSize ){
		printf("DF range exceeded\n");
		return;
	  }
	  qdf[0] = index_df_counter[linearQ[start]];//getDf(index->pointers, queries[qindex][0]);
	  unsigned int minimumDf = qdf[0];
	  for(i = 1; i < qlen; i++) {
	  	if(start+i > end){
			printf("out of range 1 \n");
			return;
	  	}
		  if (linearQ[start+i]>= index_df_vocabSize ){
			printf("DF range exceeded - Inside Loop - Not possible!\n");
			return;
		  }
		qdf[i] = index_df_counter[linearQ[start+i]];//getDf(index->pointers, queries[qindex][i]);
		if(qdf[i] < minimumDf) {
			  minimumDf = qdf[i];
		}
	  }	
	
	  // Sort query terms w.r.t. df
	  for(i = 0; i < qlen; i++) {
		unsigned int minDf = 0xFFFFFFFF;
		for(j = 0; j < qlen; j++) {
		  if(qdf[j] < minDf) {
			minDf = qdf[j];
			sortedDfIndex[i] = j;
		  }
		}
		qdf[sortedDfIndex[i]] = 0xFFFFFFFF;
	  }
	
	  for(i = 0; i < qlen; i++) {
	  	if(start+sortedDfIndex[i] > end){
			printf("out of range 2\n");
			return;
	  	}
		if (linearQ[start+sortedDfIndex[i]]>= index_pointer_vocabSize){
			printf("Pointer range exceeded - Inside Second Loop - Not possible!\n");
			return;
		}
		qStartPointers[i] = index_pointer_counter[linearQ[start+sortedDfIndex[i]]]; //getStartPointer(index->pointers, queries[qindex][sortedDfIndex[i]]);
		if (linearQ[start+sortedDfIndex[i]]>= index_df_vocabSize ){
			printf("DF range exceeded - Inside Second Loop - Not possible!\n");
			return;
		}
		qdf[i] = index_df_counter[linearQ[start+sortedDfIndex[i]]];
		//qdf[i] = getDf(index->pointers, queries[qindex][sortedDfIndex[i]]);
	  }
	
	  // Compute intersection set (or in disjunctive mode, top-k)
	  int* set;	  
	  int hits = minimumDf;
	  set = intersectSvS_GPU(index_pool_firstseg, qStartPointers, qlen, minimumDf);
	  	
	  // If output is specified, write the retrieved set to output
	  /*if(outputPath) {
		printf("Output\n");
		for(i = 0; i < hits && set[i] != TERMINAL_DOCID; i++) {
		  fprintf(fp, "q: %d no: %u\n", id, set[i]);
		}
	  } else {*/
		for(i = 0; i < hits && set[i] != TERMINAL_DOCID; i++) {
			printf("q: %d no: %u\n", id, set[i]);
		}
	  //}
	
	  // Free the allocated memory
	  free(set);
	  free(qdf);
	  free(sortedDfIndex);
	  free(qStartPointers);
}

void SvS_GPU_Entry(
	FixedIntCounter* queryLength, 
	FixedIntCounter* idToIndexMap, 
	char* outputPath, 
	InvertedIndex* index, 
	FILE * fp,
	int totalQuery,
	unsigned int* linearQ,
	int* linearQ_count,
	int tt){
	
	int i, j;
	int id = -1;
	int fqlen, pos, termid;	
	int hits = 1000;
	Algorithm algorithm = SVS;

	//printf("INside\n");
	if(queryLength==NULL || idToIndexMap == NULL || outputPath == NULL || queries == NULL || index == NULL || fp == NULL){
		printf("NULLL\n");
	}	

	fprintf(stderr, "Start SvS Data Transfer\n");

	struct timeval transferstart, transferend, gpustart, gpuend;
	gettimeofday(&transferstart, NULL);
	int* queryLength_counter;
	int* idToIndexMap_counter;
	int* index_df_counter;
	long* index_pointer_counter;
	int* index_pool_firstseg;
	unsigned int* linearQ_cuda;
	int* linearQ_count_cuda;

	hipMalloc((void**)&(queryLength_counter), 32768*sizeof(int));
	hipMalloc((void**)&(idToIndexMap_counter), 32768*sizeof(int));
	hipMalloc((void**)&(index_df_counter), DEFAULT_VOCAB_SIZE*sizeof(int));
	hipMalloc((void**)&(index_pointer_counter), DEFAULT_VOCAB_SIZE*sizeof(long));
	hipMalloc((void**)&(index_pool_firstseg), index->pool->offset*sizeof(int));	
	hipMalloc((void**)&(linearQ_cuda), tt*sizeof(unsigned int));
	hipMalloc((void**)&(linearQ_count_cuda), totalQuery*sizeof(int));

	hipMemcpy(queryLength_counter, queryLength->counter, 32768*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(idToIndexMap_counter, idToIndexMap->counter, 32768*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(index_df_counter, index->pointers->df->counter, DEFAULT_VOCAB_SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(index_pointer_counter, index->pointers->startPointers->counter, DEFAULT_VOCAB_SIZE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(index_pool_firstseg, index->pool->pool[0], index->pool->offset*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(linearQ_cuda, linearQ, tt*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(linearQ_count_cuda, linearQ_count, totalQuery*sizeof(int), hipMemcpyHostToDevice);

	gettimeofday(&transferend, NULL);

	gettimeofday(&gpustart, NULL);
	dim3  block(THREADS_PER_BLOCK, 1);
	dim3  grid((totalQuery + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK, 1);

	SvS_GPU<<<grid, block>>>(	
		queryLength_counter,
		queryLength->vocabSize,//queryLength_vocabSize,
		queryLength->defaultValue,//queryLength_defaultValue,	
		idToIndexMap_counter,
		idToIndexMap->vocabSize,
		idToIndexMap->defaultValue,		
		index_df_counter,
		index->pointers->df->vocabSize,//_df_vocabSize,
		index->pointers->df->defaultValue,		
		index_pointer_counter,
		index->pointers->startPointers->vocabSize,
		index->pointers->startPointers->defaultValue,
		index_pool_firstseg, //index->pool->pool[0]
		index->pool->offset,
		index->pool->segment,
		linearQ_cuda,
		linearQ_count_cuda,
		totalQuery);

	gettimeofday(&gpuend, NULL);

	printf("Transfer Timing: %10.0f\n",
		   ((float) ((transferend.tv_sec * 1000000 + transferend.tv_usec) -
					 (transferstart.tv_sec * 1000000 + transferstart.tv_usec))));
	printf("GPU Timing: %10.0f\n",
		   ((float) ((gpuend.tv_sec * 1000000 + gpuend.tv_usec) -
					 (gpustart.tv_sec * 1000000 + gpustart.tv_usec))));

}



