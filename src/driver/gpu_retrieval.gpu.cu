#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include "pfordelta/opt_p4.h"
#include "dictionary/Dictionary.h"
#include "buffer/FixedIntCounter.h"
#include "buffer/FixedLongCounter.h"
#include "util/ParseCommandLine.h"
#include "PostingsPool.h"
#include "Pointers.h"
#include "Config.h"
#include "InvertedIndex.h"
#include "intersection/SvS.h"
#include "intersection/WAND.h"


#ifndef RETRIEVAL_ALGO_ENUM_GUARD
#define RETRIEVAL_ALGO_ENUM_GUARD
/*typedef enum Algorithm Algorithm;
//enum Algorithm {
  SVS = 0,
  WAND = 1
};*/
#endif

#define THREADS_PER_BLOCK 512 
#define THREADS_PER_BLOCK_GLOBALPAIRS 64
#define LINEARBLOCK 100



/*************************************************************/
/* for fast unpacking of integers of fixed bit length */
/*************************************************************/

__device__ void cunpack0(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i++)  p[i] = 0;
}


__device__ void cunpack1(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 1)
  {
    p[0] = (w[0] >> 31);
    p[1] = (w[0] >> 30) & 1;
    p[2] = (w[0] >> 29) & 1;
    p[3] = (w[0] >> 28) & 1;
    p[4] = (w[0] >> 27) & 1;
    p[5] = (w[0] >> 26) & 1;
    p[6] = (w[0] >> 25) & 1;
    p[7] = (w[0] >> 24) & 1;
    p[8] = (w[0] >> 23) & 1;
    p[9] = (w[0] >> 22) & 1;
    p[10] = (w[0] >> 21) & 1;
    p[11] = (w[0] >> 20) & 1;
    p[12] = (w[0] >> 19) & 1;
    p[13] = (w[0] >> 18) & 1;
    p[14] = (w[0] >> 17) & 1;
    p[15] = (w[0] >> 16) & 1;
    p[16] = (w[0] >> 15) & 1;
    p[17] = (w[0] >> 14) & 1;
    p[18] = (w[0] >> 13) & 1;
    p[19] = (w[0] >> 12) & 1;
    p[20] = (w[0] >> 11) & 1;
    p[21] = (w[0] >> 10) & 1;
    p[22] = (w[0] >> 9) & 1;
    p[23] = (w[0] >> 8) & 1;
    p[24] = (w[0] >> 7) & 1;
    p[25] = (w[0] >> 6) & 1;
    p[26] = (w[0] >> 5) & 1;
    p[27] = (w[0] >> 4) & 1;
    p[28] = (w[0] >> 3) & 1;
    p[29] = (w[0] >> 2) & 1;
    p[30] = (w[0] >> 1) & 1;
    p[31] = (w[0]) & 1;
  }
}


__device__ void cunpack2(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 2)
  {
    p[0] = (w[0] >> 30);
    p[1] = (w[0] >> 28) & 3;
    p[2] = (w[0] >> 26) & 3;
    p[3] = (w[0] >> 24) & 3;
    p[4] = (w[0] >> 22) & 3;
    p[5] = (w[0] >> 20) & 3;
    p[6] = (w[0] >> 18) & 3;
    p[7] = (w[0] >> 16) & 3;
    p[8] = (w[0] >> 14) & 3;
    p[9] = (w[0] >> 12) & 3;
    p[10] = (w[0] >> 10) & 3;
    p[11] = (w[0] >> 8) & 3;
    p[12] = (w[0] >> 6) & 3;
    p[13] = (w[0] >> 4) & 3;
    p[14] = (w[0] >> 2) & 3;
    p[15] = (w[0]) & 3;
    p[16] = (w[1] >> 30);
    p[17] = (w[1] >> 28) & 3;
    p[18] = (w[1] >> 26) & 3;
    p[19] = (w[1] >> 24) & 3;
    p[20] = (w[1] >> 22) & 3;
    p[21] = (w[1] >> 20) & 3;
    p[22] = (w[1] >> 18) & 3;
    p[23] = (w[1] >> 16) & 3;
    p[24] = (w[1] >> 14) & 3;
    p[25] = (w[1] >> 12) & 3;
    p[26] = (w[1] >> 10) & 3;
    p[27] = (w[1] >> 8) & 3;
    p[28] = (w[1] >> 6) & 3;
    p[29] = (w[1] >> 4) & 3;
    p[30] = (w[1] >> 2) & 3;
    p[31] = (w[1]) & 3;
  }
}


__device__ void cunpack3(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 3)
  {
    p[0] = (w[0] >> 29);
    p[1] = (w[0] >> 26) & 7;
    p[2] = (w[0] >> 23) & 7;
    p[3] = (w[0] >> 20) & 7;
    p[4] = (w[0] >> 17) & 7;
    p[5] = (w[0] >> 14) & 7;
    p[6] = (w[0] >> 11) & 7;
    p[7] = (w[0] >> 8) & 7;
    p[8] = (w[0] >> 5) & 7;
    p[9] = (w[0] >> 2) & 7;
    p[10] = (w[0] << 1) & 7;
    p[10] |= (w[1] >> 31);
    p[11] = (w[1] >> 28) & 7;
    p[12] = (w[1] >> 25) & 7;
    p[13] = (w[1] >> 22) & 7;
    p[14] = (w[1] >> 19) & 7;
    p[15] = (w[1] >> 16) & 7;
    p[16] = (w[1] >> 13) & 7;
    p[17] = (w[1] >> 10) & 7;
    p[18] = (w[1] >> 7) & 7;
    p[19] = (w[1] >> 4) & 7;
    p[20] = (w[1] >> 1) & 7;
    p[21] = (w[1] << 2) & 7;
    p[21] |= (w[2] >> 30);
    p[22] = (w[2] >> 27) & 7;
    p[23] = (w[2] >> 24) & 7;
    p[24] = (w[2] >> 21) & 7;
    p[25] = (w[2] >> 18) & 7;
    p[26] = (w[2] >> 15) & 7;
    p[27] = (w[2] >> 12) & 7;
    p[28] = (w[2] >> 9) & 7;
    p[29] = (w[2] >> 6) & 7;
    p[30] = (w[2] >> 3) & 7;
    p[31] = (w[2]) & 7;
  }
}


__device__ void cunpack4(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 4)
  {
    p[0] = (w[0] >> 28);
    p[1] = (w[0] >> 24) & 15;
    p[2] = (w[0] >> 20) & 15;
    p[3] = (w[0] >> 16) & 15;
    p[4] = (w[0] >> 12) & 15;
    p[5] = (w[0] >> 8) & 15;
    p[6] = (w[0] >> 4) & 15;
    p[7] = (w[0]) & 15;
    p[8] = (w[1] >> 28);
    p[9] = (w[1] >> 24) & 15;
    p[10] = (w[1] >> 20) & 15;
    p[11] = (w[1] >> 16) & 15;
    p[12] = (w[1] >> 12) & 15;
    p[13] = (w[1] >> 8) & 15;
    p[14] = (w[1] >> 4) & 15;
    p[15] = (w[1]) & 15;
    p[16] = (w[2] >> 28);
    p[17] = (w[2] >> 24) & 15;
    p[18] = (w[2] >> 20) & 15;
    p[19] = (w[2] >> 16) & 15;
    p[20] = (w[2] >> 12) & 15;
    p[21] = (w[2] >> 8) & 15;
    p[22] = (w[2] >> 4) & 15;
    p[23] = (w[2]) & 15;
    p[24] = (w[3] >> 28);
    p[25] = (w[3] >> 24) & 15;
    p[26] = (w[3] >> 20) & 15;
    p[27] = (w[3] >> 16) & 15;
    p[28] = (w[3] >> 12) & 15;
    p[29] = (w[3] >> 8) & 15;
    p[30] = (w[3] >> 4) & 15;
    p[31] = (w[3]) & 15;
  }
}


__device__ void cunpack5(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 5)
  {
    p[0] = (w[0] >> 27);
    p[1] = (w[0] >> 22) & 31;
    p[2] = (w[0] >> 17) & 31;
    p[3] = (w[0] >> 12) & 31;
    p[4] = (w[0] >> 7) & 31;
    p[5] = (w[0] >> 2) & 31;
    p[6] = (w[0] << 3) & 31;
    p[6] |= (w[1] >> 29);
    p[7] = (w[1] >> 24) & 31;
    p[8] = (w[1] >> 19) & 31;
    p[9] = (w[1] >> 14) & 31;
    p[10] = (w[1] >> 9) & 31;
    p[11] = (w[1] >> 4) & 31;
    p[12] = (w[1] << 1) & 31;
    p[12] |= (w[2] >> 31);
    p[13] = (w[2] >> 26) & 31;
    p[14] = (w[2] >> 21) & 31;
    p[15] = (w[2] >> 16) & 31;
    p[16] = (w[2] >> 11) & 31;
    p[17] = (w[2] >> 6) & 31;
    p[18] = (w[2] >> 1) & 31;
    p[19] = (w[2] << 4) & 31;
    p[19] |= (w[3] >> 28);
    p[20] = (w[3] >> 23) & 31;
    p[21] = (w[3] >> 18) & 31;
    p[22] = (w[3] >> 13) & 31;
    p[23] = (w[3] >> 8) & 31;
    p[24] = (w[3] >> 3) & 31;
    p[25] = (w[3] << 2) & 31;
    p[25] |= (w[4] >> 30);
    p[26] = (w[4] >> 25) & 31;
    p[27] = (w[4] >> 20) & 31;
    p[28] = (w[4] >> 15) & 31;
    p[29] = (w[4] >> 10) & 31;
    p[30] = (w[4] >> 5) & 31;
    p[31] = (w[4]) & 31;
  }
}


__device__ void cunpack6(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 6)
  {
    p[0] = (w[0] >> 26);
    p[1] = (w[0] >> 20) & 63;
    p[2] = (w[0] >> 14) & 63;
    p[3] = (w[0] >> 8) & 63;
    p[4] = (w[0] >> 2) & 63;
    p[5] = (w[0] << 4) & 63;
    p[5] |= (w[1] >> 28);
    p[6] = (w[1] >> 22) & 63;
    p[7] = (w[1] >> 16) & 63;
    p[8] = (w[1] >> 10) & 63;
    p[9] = (w[1] >> 4) & 63;
    p[10] = (w[1] << 2) & 63;
    p[10] |= (w[2] >> 30);
    p[11] = (w[2] >> 24) & 63;
    p[12] = (w[2] >> 18) & 63;
    p[13] = (w[2] >> 12) & 63;
    p[14] = (w[2] >> 6) & 63;
    p[15] = (w[2]) & 63;
    p[16] = (w[3] >> 26);
    p[17] = (w[3] >> 20) & 63;
    p[18] = (w[3] >> 14) & 63;
    p[19] = (w[3] >> 8) & 63;
    p[20] = (w[3] >> 2) & 63;
    p[21] = (w[3] << 4) & 63;
    p[21] |= (w[4] >> 28);
    p[22] = (w[4] >> 22) & 63;
    p[23] = (w[4] >> 16) & 63;
    p[24] = (w[4] >> 10) & 63;
    p[25] = (w[4] >> 4) & 63;
    p[26] = (w[4] << 2) & 63;
    p[26] |= (w[5] >> 30);
    p[27] = (w[5] >> 24) & 63;
    p[28] = (w[5] >> 18) & 63;
    p[29] = (w[5] >> 12) & 63;
    p[30] = (w[5] >> 6) & 63;
    p[31] = (w[5]) & 63;
  }
}


__device__ void cunpack7(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 7)
  {
    p[0] = (w[0] >> 25);
    p[1] = (w[0] >> 18) & 127;
    p[2] = (w[0] >> 11) & 127;
    p[3] = (w[0] >> 4) & 127;
    p[4] = (w[0] << 3) & 127;
    p[4] |= (w[1] >> 29);
    p[5] = (w[1] >> 22) & 127;
    p[6] = (w[1] >> 15) & 127;
    p[7] = (w[1] >> 8) & 127;
    p[8] = (w[1] >> 1) & 127;
    p[9] = (w[1] << 6) & 127;
    p[9] |= (w[2] >> 26);
    p[10] = (w[2] >> 19) & 127;
    p[11] = (w[2] >> 12) & 127;
    p[12] = (w[2] >> 5) & 127;
    p[13] = (w[2] << 2) & 127;
    p[13] |= (w[3] >> 30);
    p[14] = (w[3] >> 23) & 127;
    p[15] = (w[3] >> 16) & 127;
    p[16] = (w[3] >> 9) & 127;
    p[17] = (w[3] >> 2) & 127;
    p[18] = (w[3] << 5) & 127;
    p[18] |= (w[4] >> 27);
    p[19] = (w[4] >> 20) & 127;
    p[20] = (w[4] >> 13) & 127;
    p[21] = (w[4] >> 6) & 127;
    p[22] = (w[4] << 1) & 127;
    p[22] |= (w[5] >> 31);
    p[23] = (w[5] >> 24) & 127;
    p[24] = (w[5] >> 17) & 127;
    p[25] = (w[5] >> 10) & 127;
    p[26] = (w[5] >> 3) & 127;
    p[27] = (w[5] << 4) & 127;
    p[27] |= (w[6] >> 28);
    p[28] = (w[6] >> 21) & 127;
    p[29] = (w[6] >> 14) & 127;
    p[30] = (w[6] >> 7) & 127;
    p[31] = (w[6]) & 127;
  }
}


__device__ void cunpack8(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 8)
  {
    p[0] = (w[0] >> 24);
    p[1] = (w[0] >> 16) & 255;
    p[2] = (w[0] >> 8) & 255;
    p[3] = (w[0]) & 255;
    p[4] = (w[1] >> 24);
    p[5] = (w[1] >> 16) & 255;
    p[6] = (w[1] >> 8) & 255;
    p[7] = (w[1]) & 255;
    p[8] = (w[2] >> 24);
    p[9] = (w[2] >> 16) & 255;
    p[10] = (w[2] >> 8) & 255;
    p[11] = (w[2]) & 255;
    p[12] = (w[3] >> 24);
    p[13] = (w[3] >> 16) & 255;
    p[14] = (w[3] >> 8) & 255;
    p[15] = (w[3]) & 255;
    p[16] = (w[4] >> 24);
    p[17] = (w[4] >> 16) & 255;
    p[18] = (w[4] >> 8) & 255;
    p[19] = (w[4]) & 255;
    p[20] = (w[5] >> 24);
    p[21] = (w[5] >> 16) & 255;
    p[22] = (w[5] >> 8) & 255;
    p[23] = (w[5]) & 255;
    p[24] = (w[6] >> 24);
    p[25] = (w[6] >> 16) & 255;
    p[26] = (w[6] >> 8) & 255;
    p[27] = (w[6]) & 255;
    p[28] = (w[7] >> 24);
    p[29] = (w[7] >> 16) & 255;
    p[30] = (w[7] >> 8) & 255;
    p[31] = (w[7]) & 255;
  }
}


__device__ void cunpack9(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 9)
  {
    p[0] = (w[0] >> 23);
    p[1] = (w[0] >> 14) & 511;
    p[2] = (w[0] >> 5) & 511;
    p[3] = (w[0] << 4) & 511;
    p[3] |= (w[1] >> 28);
    p[4] = (w[1] >> 19) & 511;
    p[5] = (w[1] >> 10) & 511;
    p[6] = (w[1] >> 1) & 511;
    p[7] = (w[1] << 8) & 511;
    p[7] |= (w[2] >> 24);
    p[8] = (w[2] >> 15) & 511;
    p[9] = (w[2] >> 6) & 511;
    p[10] = (w[2] << 3) & 511;
    p[10] |= (w[3] >> 29);
    p[11] = (w[3] >> 20) & 511;
    p[12] = (w[3] >> 11) & 511;
    p[13] = (w[3] >> 2) & 511;
    p[14] = (w[3] << 7) & 511;
    p[14] |= (w[4] >> 25);
    p[15] = (w[4] >> 16) & 511;
    p[16] = (w[4] >> 7) & 511;
    p[17] = (w[4] << 2) & 511;
    p[17] |= (w[5] >> 30);
    p[18] = (w[5] >> 21) & 511;
    p[19] = (w[5] >> 12) & 511;
    p[20] = (w[5] >> 3) & 511;
    p[21] = (w[5] << 6) & 511;
    p[21] |= (w[6] >> 26);
    p[22] = (w[6] >> 17) & 511;
    p[23] = (w[6] >> 8) & 511;
    p[24] = (w[6] << 1) & 511;
    p[24] |= (w[7] >> 31);
    p[25] = (w[7] >> 22) & 511;
    p[26] = (w[7] >> 13) & 511;
    p[27] = (w[7] >> 4) & 511;
    p[28] = (w[7] << 5) & 511;
    p[28] |= (w[8] >> 27);
    p[29] = (w[8] >> 18) & 511;
    p[30] = (w[8] >> 9) & 511;
    p[31] = (w[8]) & 511;
  }
}


__device__ void cunpack10(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 10)
  {
    p[0] = (w[0] >> 22);
    p[1] = (w[0] >> 12) & 1023;
    p[2] = (w[0] >> 2) & 1023;
    p[3] = (w[0] << 8) & 1023;
    p[3] |= (w[1] >> 24);
    p[4] = (w[1] >> 14) & 1023;
    p[5] = (w[1] >> 4) & 1023;
    p[6] = (w[1] << 6) & 1023;
    p[6] |= (w[2] >> 26);
    p[7] = (w[2] >> 16) & 1023;
    p[8] = (w[2] >> 6) & 1023;
    p[9] = (w[2] << 4) & 1023;
    p[9] |= (w[3] >> 28);
    p[10] = (w[3] >> 18) & 1023;
    p[11] = (w[3] >> 8) & 1023;
    p[12] = (w[3] << 2) & 1023;
    p[12] |= (w[4] >> 30);
    p[13] = (w[4] >> 20) & 1023;
    p[14] = (w[4] >> 10) & 1023;
    p[15] = (w[4]) & 1023;
    p[16] = (w[5] >> 22);
    p[17] = (w[5] >> 12) & 1023;
    p[18] = (w[5] >> 2) & 1023;
    p[19] = (w[5] << 8) & 1023;
    p[19] |= (w[6] >> 24);
    p[20] = (w[6] >> 14) & 1023;
    p[21] = (w[6] >> 4) & 1023;
    p[22] = (w[6] << 6) & 1023;
    p[22] |= (w[7] >> 26);
    p[23] = (w[7] >> 16) & 1023;
    p[24] = (w[7] >> 6) & 1023;
    p[25] = (w[7] << 4) & 1023;
    p[25] |= (w[8] >> 28);
    p[26] = (w[8] >> 18) & 1023;
    p[27] = (w[8] >> 8) & 1023;
    p[28] = (w[8] << 2) & 1023;
    p[28] |= (w[9] >> 30);
    p[29] = (w[9] >> 20) & 1023;
    p[30] = (w[9] >> 10) & 1023;
    p[31] = (w[9]) & 1023;
  }
}


__device__ void cunpack11(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 11)
  {
    p[0] = (w[0] >> 21);
    p[1] = (w[0] >> 10) & 2047;
    p[2] = (w[0] << 1) & 2047;
    p[2] |= (w[1] >> 31);
    p[3] = (w[1] >> 20) & 2047;
    p[4] = (w[1] >> 9) & 2047;
    p[5] = (w[1] << 2) & 2047;
    p[5] |= (w[2] >> 30);
    p[6] = (w[2] >> 19) & 2047;
    p[7] = (w[2] >> 8) & 2047;
    p[8] = (w[2] << 3) & 2047;
    p[8] |= (w[3] >> 29);
    p[9] = (w[3] >> 18) & 2047;
    p[10] = (w[3] >> 7) & 2047;
    p[11] = (w[3] << 4) & 2047;
    p[11] |= (w[4] >> 28);
    p[12] = (w[4] >> 17) & 2047;
    p[13] = (w[4] >> 6) & 2047;
    p[14] = (w[4] << 5) & 2047;
    p[14] |= (w[5] >> 27);
    p[15] = (w[5] >> 16) & 2047;
    p[16] = (w[5] >> 5) & 2047;
    p[17] = (w[5] << 6) & 2047;
    p[17] |= (w[6] >> 26);
    p[18] = (w[6] >> 15) & 2047;
    p[19] = (w[6] >> 4) & 2047;
    p[20] = (w[6] << 7) & 2047;
    p[20] |= (w[7] >> 25);
    p[21] = (w[7] >> 14) & 2047;
    p[22] = (w[7] >> 3) & 2047;
    p[23] = (w[7] << 8) & 2047;
    p[23] |= (w[8] >> 24);
    p[24] = (w[8] >> 13) & 2047;
    p[25] = (w[8] >> 2) & 2047;
    p[26] = (w[8] << 9) & 2047;
    p[26] |= (w[9] >> 23);
    p[27] = (w[9] >> 12) & 2047;
    p[28] = (w[9] >> 1) & 2047;
    p[29] = (w[9] << 10) & 2047;
    p[29] |= (w[10] >> 22);
    p[30] = (w[10] >> 11) & 2047;
    p[31] = (w[10]) & 2047;
  }
}


__device__ void cunpack12(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 12)
  {
    p[0] = (w[0] >> 20);
    p[1] = (w[0] >> 8) & 4095;
    p[2] = (w[0] << 4) & 4095;
    p[2] |= (w[1] >> 28);
    p[3] = (w[1] >> 16) & 4095;
    p[4] = (w[1] >> 4) & 4095;
    p[5] = (w[1] << 8) & 4095;
    p[5] |= (w[2] >> 24);
    p[6] = (w[2] >> 12) & 4095;
    p[7] = (w[2]) & 4095;
    p[8] = (w[3] >> 20);
    p[9] = (w[3] >> 8) & 4095;
    p[10] = (w[3] << 4) & 4095;
    p[10] |= (w[4] >> 28);
    p[11] = (w[4] >> 16) & 4095;
    p[12] = (w[4] >> 4) & 4095;
    p[13] = (w[4] << 8) & 4095;
    p[13] |= (w[5] >> 24);
    p[14] = (w[5] >> 12) & 4095;
    p[15] = (w[5]) & 4095;
    p[16] = (w[6] >> 20);
    p[17] = (w[6] >> 8) & 4095;
    p[18] = (w[6] << 4) & 4095;
    p[18] |= (w[7] >> 28);
    p[19] = (w[7] >> 16) & 4095;
    p[20] = (w[7] >> 4) & 4095;
    p[21] = (w[7] << 8) & 4095;
    p[21] |= (w[8] >> 24);
    p[22] = (w[8] >> 12) & 4095;
    p[23] = (w[8]) & 4095;
    p[24] = (w[9] >> 20);
    p[25] = (w[9] >> 8) & 4095;
    p[26] = (w[9] << 4) & 4095;
    p[26] |= (w[10] >> 28);
    p[27] = (w[10] >> 16) & 4095;
    p[28] = (w[10] >> 4) & 4095;
    p[29] = (w[10] << 8) & 4095;
    p[29] |= (w[11] >> 24);
    p[30] = (w[11] >> 12) & 4095;
    p[31] = (w[11]) & 4095;
  }
}


__device__ void cunpack13(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 13)
  {
    p[0] = (w[0] >> 19);
    p[1] = (w[0] >> 6) & 8191;
    p[2] = (w[0] << 7) & 8191;
    p[2] |= (w[1] >> 25);
    p[3] = (w[1] >> 12) & 8191;
    p[4] = (w[1] << 1) & 8191;
    p[4] |= (w[2] >> 31);
    p[5] = (w[2] >> 18) & 8191;
    p[6] = (w[2] >> 5) & 8191;
    p[7] = (w[2] << 8) & 8191;
    p[7] |= (w[3] >> 24);
    p[8] = (w[3] >> 11) & 8191;
    p[9] = (w[3] << 2) & 8191;
    p[9] |= (w[4] >> 30);
    p[10] = (w[4] >> 17) & 8191;
    p[11] = (w[4] >> 4) & 8191;
    p[12] = (w[4] << 9) & 8191;
    p[12] |= (w[5] >> 23);
    p[13] = (w[5] >> 10) & 8191;
    p[14] = (w[5] << 3) & 8191;
    p[14] |= (w[6] >> 29);
    p[15] = (w[6] >> 16) & 8191;
    p[16] = (w[6] >> 3) & 8191;
    p[17] = (w[6] << 10) & 8191;
    p[17] |= (w[7] >> 22);
    p[18] = (w[7] >> 9) & 8191;
    p[19] = (w[7] << 4) & 8191;
    p[19] |= (w[8] >> 28);
    p[20] = (w[8] >> 15) & 8191;
    p[21] = (w[8] >> 2) & 8191;
    p[22] = (w[8] << 11) & 8191;
    p[22] |= (w[9] >> 21);
    p[23] = (w[9] >> 8) & 8191;
    p[24] = (w[9] << 5) & 8191;
    p[24] |= (w[10] >> 27);
    p[25] = (w[10] >> 14) & 8191;
    p[26] = (w[10] >> 1) & 8191;
    p[27] = (w[10] << 12) & 8191;
    p[27] |= (w[11] >> 20);
    p[28] = (w[11] >> 7) & 8191;
    p[29] = (w[11] << 6) & 8191;
    p[29] |= (w[12] >> 26);
    p[30] = (w[12] >> 13) & 8191;
    p[31] = (w[12]) & 8191;
  }
}


__device__ void cunpack16(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 16)
  {
    p[0] = (w[0] >> 16);
    p[1] = (w[0]) & 65535;
    p[2] = (w[1] >> 16);
    p[3] = (w[1]) & 65535;
    p[4] = (w[2] >> 16);
    p[5] = (w[2]) & 65535;
    p[6] = (w[3] >> 16);
    p[7] = (w[3]) & 65535;
    p[8] = (w[4] >> 16);
    p[9] = (w[4]) & 65535;
    p[10] = (w[5] >> 16);
    p[11] = (w[5]) & 65535;
    p[12] = (w[6] >> 16);
    p[13] = (w[6]) & 65535;
    p[14] = (w[7] >> 16);
    p[15] = (w[7]) & 65535;
    p[16] = (w[8] >> 16);
    p[17] = (w[8]) & 65535;
    p[18] = (w[9] >> 16);
    p[19] = (w[9]) & 65535;
    p[20] = (w[10] >> 16);
    p[21] = (w[10]) & 65535;
    p[22] = (w[11] >> 16);
    p[23] = (w[11]) & 65535;
    p[24] = (w[12] >> 16);
    p[25] = (w[12]) & 65535;
    p[26] = (w[13] >> 16);
    p[27] = (w[13]) & 65535;
    p[28] = (w[14] >> 16);
    p[29] = (w[14]) & 65535;
    p[30] = (w[15] >> 16);
    p[31] = (w[15]) & 65535;
  }
}


__device__ void cunpack20(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 20)
  {
    p[0] = (w[0] >> 12);
    p[1] = (w[0] << 8) & ((1<<20)-1);
    p[1] |= (w[1] >> 24);
    p[2] = (w[1] >> 4) & ((1<<20)-1);
    p[3] = (w[1] << 16) & ((1<<20)-1);
    p[3] |= (w[2] >> 16);
    p[4] = (w[2] << 4) & ((1<<20)-1);
    p[4] |= (w[3] >> 28);
    p[5] = (w[3] >> 8) & ((1<<20)-1);
    p[6] = (w[3] << 12) & ((1<<20)-1);
    p[6] |= (w[4] >> 20);
    p[7] = (w[4]) & ((1<<20)-1);
    p[8] = (w[5] >> 12);
    p[9] = (w[5] << 8) & ((1<<20)-1);
    p[9] |= (w[6] >> 24);
    p[10] = (w[6] >> 4) & ((1<<20)-1);
    p[11] = (w[6] << 16) & ((1<<20)-1);
    p[11] |= (w[7] >> 16);
    p[12] = (w[7] << 4) & ((1<<20)-1);
    p[12] |= (w[8] >> 28);
    p[13] = (w[8] >> 8) & ((1<<20)-1);
    p[14] = (w[8] << 12) & ((1<<20)-1);
    p[14] |= (w[9] >> 20);
    p[15] = (w[9]) & ((1<<20)-1);
    p[16] = (w[10] >> 12);
    p[17] = (w[10] << 8) & ((1<<20)-1);
    p[17] |= (w[11] >> 24);
    p[18] = (w[11] >> 4) & ((1<<20)-1);
    p[19] = (w[11] << 16) & ((1<<20)-1);
    p[19] |= (w[12] >> 16);
    p[20] = (w[12] << 4) & ((1<<20)-1);
    p[20] |= (w[13] >> 28);
    p[21] = (w[13] >> 8) & ((1<<20)-1);
    p[22] = (w[13] << 12) & ((1<<20)-1);
    p[22] |= (w[14] >> 20);
    p[23] = (w[14]) & ((1<<20)-1);
    p[24] = (w[15] >> 12);
    p[25] = (w[15] << 8) & ((1<<20)-1);
    p[25] |= (w[16] >> 24);
    p[26] = (w[16] >> 4) & ((1<<20)-1);
    p[27] = (w[16] << 16) & ((1<<20)-1);
    p[27] |= (w[17] >> 16);
    p[28] = (w[17] << 4) & ((1<<20)-1);
    p[28] |= (w[18] >> 28);
    p[29] = (w[18] >> 8) & ((1<<20)-1);
    p[30] = (w[18] << 12) & ((1<<20)-1);
    p[30] |= (w[19] >> 20);
    p[31] = (w[19]) & ((1<<20)-1);
  }
}


__device__ void cunpack32(unsigned int *p, unsigned int *w)
{
  int i;

  for (i = 0; i < BLOCK_SIZE; i += 32, p += 32, w += 32)
  {
    p[0] = w[0];
    p[1] = w[1];
    p[2] = w[2];
    p[3] = w[3];
    p[4] = w[4];
    p[5] = w[5];
    p[6] = w[6];
    p[7] = w[7];
    p[8] = w[8];
    p[9] = w[9];
    p[10] = w[10];
    p[11] = w[11];
    p[12] = w[12];
    p[13] = w[13];
    p[14] = w[14];
    p[15] = w[15];
    p[16] = w[16];
    p[17] = w[17];
    p[18] = w[18];
    p[19] = w[19];
    p[20] = w[20];
    p[21] = w[21];
    p[22] = w[22];
    p[23] = w[23];
    p[24] = w[24];
    p[25] = w[25];
    p[26] = w[26];
    p[27] = w[27];
    p[28] = w[28];
    p[29] = w[29];
    p[30] = w[30];
    p[31] = w[31];
  }
}

/*modified p4decode */
__device__ unsigned int *detailed_p4_decode_new(unsigned int *_p, unsigned int *_w,  unsigned int * all_array, int delta)
{

  int i, s;
  unsigned int x;
  int flag = _w[0];
  (_w)++;

  unsigned int *_ww,*_pp;
  unsigned int b = ((flag>>10) & 31);
  unsigned int e_n = (flag & 1023) ;

  //(unpack[b])(_p, _w);

  if(b <= 13 ){
	b = (int)b;
  }else if (b == 14){
    b = 16;
  } else if (b == 15){
    b = 20;
  } else if (b == 16) {
    b = 32;
  }
  
  switch(b) { 
	case 0: cunpack0(_p, _w); break;
	case 1: cunpack1(_p, _w); break;
	case 2: cunpack2(_p, _w);break;
	case 3: cunpack3(_p, _w);break;
	case 4: cunpack4(_p, _w);break;
	case 5: cunpack5(_p, _w);break;
	case 6: cunpack6(_p, _w);break;
	case 7: cunpack7(_p, _w);break;
	case 8: cunpack8(_p, _w);break;
	case 9: cunpack9(_p, _w);break;
	case 10: cunpack10(_p, _w);break;
	case 11: cunpack11(_p, _w);break;
	case 12: cunpack12(_p, _w);break;
	case 13: cunpack13(_p, _w);break;
	case 16: cunpack16(_p, _w);break;
	case 20: cunpack20(_p, _w);break;
	case 32: cunpack32(_p, _w);break; 
  }

  //b = cnum[b];
  _w += ((b * BLOCK_SIZE)>>5);
  unsigned int _k = 0;
  unsigned int psum = 0;
  if(e_n != 0 )
  {
    for (_pp = all_array, _ww = (unsigned int *)(_w); _pp < &(all_array[e_n*2]);)
    {
      S16_DECODE(_ww, _pp);
    }

    _w += (_ww - _w);
    psum = all_array[0];

    for(i=0;i<e_n;i++)
    {
      _p[psum] += (all_array[e_n+i]<<b);
      psum += all_array[ i + 1] + 1;
    }
  }

  if(delta) {
    for(i = 1; i < BLOCK_SIZE && _p[i] != 0; i++) {
      _p[i] += _p[i - 1];
    }
  }

  return(_w);
}

__device__ int decompressDocidBlock_GPU(int* pool, unsigned int* outBlock, long pointer) {
  int pSegment = DECODE_SEGMENT(pointer);
  unsigned int pOffset = DECODE_OFFSET(pointer);

  unsigned int aux[BLOCK_SIZE*4];
  unsigned int* block = (unsigned int*) &pool[pOffset + 5];
  detailed_p4_decode_new(outBlock, block, aux, 1);

  return pool[pOffset + 3];
}

__device__ long nextPointer_GPU(int* pool, long pointer) {
  if(pointer == UNDEFINED_POINTER) {
    return UNDEFINED_POINTER;
  }
  int pSegment = DECODE_SEGMENT(pointer);
  unsigned int pOffset = DECODE_OFFSET(pointer);

  if(pool[pOffset + 1] == UNKNOWN_SEGMENT) {
    return UNDEFINED_POINTER;
  }

  return ENCODE_POINTER(pool[pOffset + 1],
                        pool[pOffset + 2]);
}

__device__ int* intersectPostingsLists_SvS_GPU(int* pool, long a, long b, int minDf) {
  int* set = (int*) malloc(minDf * sizeof(int));
  memset(set, 0, minDf * sizeof(int));

  unsigned int* dataA = (unsigned int*) malloc(BLOCK_SIZE * 2 * sizeof(unsigned int));
  unsigned int* dataB = (unsigned int*) malloc(BLOCK_SIZE * 2 * sizeof(unsigned int));
  memset(dataA, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
  memset(dataB, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));

  int cA = decompressDocidBlock_GPU(pool, dataA, a);
  int cB = decompressDocidBlock_GPU(pool, dataB, b);
  int iSet = 0, iA = 0, iB = 0;

  while(a != UNDEFINED_POINTER && b != UNDEFINED_POINTER) {
    if(dataB[iB] == dataA[iA]) {
      set[iSet++] = dataA[iA];
      iA++;
      iB++;
    }

    if(iA == cA) {
      a = nextPointer_GPU(pool, a);
      if(a == UNDEFINED_POINTER) {
        break;
      }
      memset(dataA, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
      cA = decompressDocidBlock_GPU(pool, dataA, a);
      iA = 0;
    }
    if(iB == cB) {
      b = nextPointer_GPU(pool, b);
      if(b == UNDEFINED_POINTER) {
        break;
      }
      memset(dataB, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
      cB = decompressDocidBlock_GPU(pool, dataB, b);
      iB = 0;
    }

    if(dataA[iA] < dataB[iB]) {
      if(dataA[cA - 1] < dataB[iB]) {
        iA = cA - 1;
      }
      while(dataA[iA] < dataB[iB]) {
        iA++;
        if(iA == cA) {
          a = nextPointer_GPU(pool, a);
          if(a == UNDEFINED_POINTER) {
            break;
          }
          memset(dataA, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
          cA = decompressDocidBlock_GPU(pool, dataA, a);
          iA = 0;
        }
        if(dataA[cA - 1] < dataB[iB]) {
          iA = cA - 1;
        }
      }
    } else {
      if(dataB[cB - 1] < dataA[iA]) {
        iB = cB - 1;
      }
      while(dataB[iB] < dataA[iA]) {
        iB++;
        if(iB == cB) {
          b = nextPointer_GPU(pool, b);
          if(b == UNDEFINED_POINTER) {
            break;
          }
          memset(dataB, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
          cB = decompressDocidBlock_GPU(pool, dataB, b);
          iB = 0;
        }
        if(dataB[cB - 1] < dataA[iA]) {
          iB = cB - 1;
        }
      }
    }
  }

  if(iSet < minDf) {
    set[iSet] = TERMINAL_DOCID;
  }

  free(dataA);
  free(dataB);

  return set;
}

__device__ int intersectSetPostingsList_SvS_GPU(int* pool, long a, int* currentSet, int len) {
  unsigned int* data = (unsigned int*) malloc(BLOCK_SIZE * 2 * sizeof(unsigned int));
  memset(data, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
  
  int c = decompressDocidBlock_GPU(pool, data, a);
  int iSet = 0, iCurrent = 0, i = 0;

  while(a != UNDEFINED_POINTER && iCurrent < len) {
    if(currentSet[iCurrent] == TERMINAL_DOCID) {
      break;
    }
    if(data[i] == currentSet[iCurrent]) {
      currentSet[iSet++] = currentSet[iCurrent];
      iCurrent++;
      i++;
    }

    if(i == c) {
      a = nextPointer_GPU(pool, a);
      if(a == UNDEFINED_POINTER) {
        break;
      }
      memset(data, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
      c = decompressDocidBlock_GPU(pool, data, a);
      i = 0;
    }
    if(iCurrent == len) {
      break;
    }
    if(currentSet[iCurrent] == TERMINAL_DOCID) {
      break;
    }

    if(data[i] < currentSet[iCurrent]) {
      if(data[c - 1] < currentSet[iCurrent]) {
        i = c - 1;
      }
      while(data[i] < currentSet[iCurrent]) {
        i++;
        if(i == c) {
          a = nextPointer_GPU(pool, a);
          if(a == UNDEFINED_POINTER) {
            break;
          }
          memset(data, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
          c = decompressDocidBlock_GPU(pool, data, a);
          i = 0;
        }
        if(data[c - 1] < currentSet[iCurrent]) {
          i = c - 1;
        }
      }
    } else {
      while(currentSet[iCurrent] < data[i]) {
        iCurrent++;
        if(iCurrent == len) {
          break;
        }
        if(currentSet[iCurrent] == TERMINAL_DOCID) {
          break;
        }
      }
    }
  }

  if(iSet < len) {
    currentSet[iSet] = TERMINAL_DOCID;
  }

  free(data);
  return iSet;
}

__device__ int* intersectSvS_GPU(int* pool, long* startPointers, int len, int minDf) {
  if(len < 2) {
    unsigned int* block = (unsigned int*) malloc(BLOCK_SIZE * 2 *sizeof(unsigned int));
    memset(block, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
    
    int* set = (int*) malloc(minDf * sizeof(int));
    memset(set, 0, minDf* sizeof(unsigned int));
    
    int iSet = 0;
    long t = startPointers[0];
    while(t != UNDEFINED_POINTER) {
      memset(block, 0, BLOCK_SIZE * 2 * sizeof(unsigned int));
      int c = decompressDocidBlock_GPU(pool, block, t);
      memcpy(&set[iSet], block, c * sizeof(int));
      iSet += c;
      t = nextPointer_GPU(pool, t);
    }
    free(block);
    return set;
  }

  int* set = intersectPostingsLists_SvS_GPU(pool, startPointers[0], startPointers[1], minDf);
  int i;
  for(i = 2; i < len; i++) {
    intersectSetPostingsList_SvS_GPU(pool, startPointers[i], set, minDf);
  }
  return set;
}

__global__ void SvS_GPU(
	int* queryLength_counter,
	unsigned int queryLength_vocabSize,
	DefaultValue queryLength_defaultValue,	
	int* idToIndexMap_counter,
	unsigned int idToIndexMap_vocabSize,
	DefaultValue idToIndexMap_defaultValue,		
	int* index_df_counter,
	unsigned int index_df_vocabSize,
	DefaultValue index_df_defaultValue,		
	long* index_pointer_counter,
	unsigned int index_pointer_vocabSize,
	DefaultValue index_pointer_defaultValue,	
	int* index_pool_firstseg, //index->pool->pool[0]
	unsigned int index_pool_offset,
	unsigned int index_pool_segment,	
	unsigned int* linearQ,
	int* linearQ_count,
	int totalQuery){

	int id = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
	if(id >= queryLength_vocabSize) {
	  return;
	}
	
	if(queryLength_counter[id] == queryLength_defaultValue) {
	  return;
	}

	 printf("Start QID = %d\n", id);
	  // Measure elapsed time
	  int i, j;
	  int qlen = queryLength_counter[id];
	  int qindex = idToIndexMap_counter[id];
	  if (qindex > totalQuery){
	  	printf("Exceed the range!\n");
		return;
	  }
	  
	  unsigned int* qdf = (unsigned int*) malloc(qlen * sizeof(unsigned int));
	  memset(qdf, 0, qlen * sizeof(unsigned int));
	  
	  int* sortedDfIndex = (int*) malloc(qlen * sizeof(int));
	  memset(sortedDfIndex, 0, qlen * sizeof(unsigned int));

	  long* qStartPointers = (long*) malloc(qlen * sizeof(long));
	  memset(qStartPointers, 0, qlen * sizeof(unsigned int));
	  
	  int end = linearQ_count[qindex];
	  int start = 0;
	  if (qindex > 0){
		start = linearQ_count[qindex-1];
	  }
	  
	  if (linearQ[start]>= index_df_vocabSize ){
		printf("DF range exceeded\n");
		return;
	  }
	  qdf[0] = index_df_counter[linearQ[start]];//getDf(index->pointers, queries[qindex][0]);
	  unsigned int minimumDf = qdf[0];
	  for(i = 1; i < qlen; i++) {
	  	if(start+i > end){
			printf("out of range 1 \n");
			return;
	  	}
		  if (linearQ[start+i]>= index_df_vocabSize ){
			printf("DF range exceeded - Inside Loop - Not possible!\n");
			return;
		  }
		qdf[i] = index_df_counter[linearQ[start+i]];//getDf(index->pointers, queries[qindex][i]);
		if(qdf[i] < minimumDf) {
			  minimumDf = qdf[i];
		}
	  }	
	
	  // Sort query terms w.r.t. df
	  for(i = 0; i < qlen; i++) {
		unsigned int minDf = 0xFFFFFFFF;
		for(j = 0; j < qlen; j++) {
		  if(qdf[j] < minDf) {
			minDf = qdf[j];
			sortedDfIndex[i] = j;
		  }
		}
		qdf[sortedDfIndex[i]] = 0xFFFFFFFF;
	  }
	
	  for(i = 0; i < qlen; i++) {
	  	if(start+sortedDfIndex[i] > end){
			printf("out of range 2\n");
			return;
	  	}
		if (linearQ[start+sortedDfIndex[i]]>= index_pointer_vocabSize){
			printf("Pointer range exceeded - Inside Second Loop - Not possible!\n");
			return;
		}
		qStartPointers[i] = index_pointer_counter[linearQ[start+sortedDfIndex[i]]]; //getStartPointer(index->pointers, queries[qindex][sortedDfIndex[i]]);
		if (linearQ[start+sortedDfIndex[i]]>= index_df_vocabSize ){
			printf("DF range exceeded - Inside Second Loop - Not possible!\n");
			return;
		}
		qdf[i] = index_df_counter[linearQ[start+sortedDfIndex[i]]];
		//qdf[i] = getDf(index->pointers, queries[qindex][sortedDfIndex[i]]);
	  }
	
	  // Compute intersection set (or in disjunctive mode, top-k)
	  int* set;	  
	  int hits = minimumDf;
	  set = intersectSvS_GPU(index_pool_firstseg, qStartPointers, qlen, minimumDf);
	  	
	  // If output is specified, write the retrieved set to output
	  /*if(outputPath) {
		printf("Output\n");
		for(i = 0; i < hits && set[i] != TERMINAL_DOCID; i++) {
		  fprintf(fp, "q: %d no: %u\n", id, set[i]);
		}
	  } else {*/
		for(i = 0; i < hits && set[i] != TERMINAL_DOCID; i++) {
			printf("q: %d no: %u\n", id, set[i]);
		}
	  //}
	
	  // Free the allocated memory
	  free(set);
	  free(qdf);
	  free(sortedDfIndex);
	  free(qStartPointers);
}

void SvS_GPU_Entry(
	FixedIntCounter* queryLength, 
	FixedIntCounter* idToIndexMap, 
	char* outputPath, 
	InvertedIndex* index, 
	FILE * fp,
	int totalQuery,
	unsigned int* linearQ,
	int* linearQ_count,
	int tt){
	
	int i, j;
	int id = -1;
	int fqlen, pos, termid;	
	int hits = 1000;
	//Algorithm algorithm = SVS;

	printf("INside SvS GPU Entry!!!\n");
	if(queryLength==NULL || idToIndexMap == NULL || index == NULL){
		printf("NULLL\n");
	}	

	fprintf(stderr, "Start SvS Data Transfer\n");

	struct timeval transferstart, transferend, gpustart, gpuend;
	gettimeofday(&transferstart, NULL);
	int* queryLength_counter;
	int* idToIndexMap_counter;
	int* index_df_counter;
	long* index_pointer_counter;
	int* index_pool_firstseg;
	unsigned int* linearQ_cuda;
	int* linearQ_count_cuda;

	hipMalloc((void**)&(queryLength_counter), 32768*sizeof(int));
	hipMalloc((void**)&(idToIndexMap_counter), 32768*sizeof(int));
	hipMalloc((void**)&(index_df_counter), DEFAULT_VOCAB_SIZE*sizeof(int));
	hipMalloc((void**)&(index_pointer_counter), DEFAULT_VOCAB_SIZE*sizeof(long));
	hipMalloc((void**)&(index_pool_firstseg), index->pool->offset*sizeof(int));	
	hipMalloc((void**)&(linearQ_cuda), tt*sizeof(unsigned int));
	hipMalloc((void**)&(linearQ_count_cuda), totalQuery*sizeof(int));

	hipMemcpy(queryLength_counter, queryLength->counter, 32768*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(idToIndexMap_counter, idToIndexMap->counter, 32768*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(index_df_counter, index->pointers->df->counter, DEFAULT_VOCAB_SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(index_pointer_counter, index->pointers->startPointers->counter, DEFAULT_VOCAB_SIZE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(index_pool_firstseg, index->pool->pool[0], index->pool->offset*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(linearQ_cuda, linearQ, tt*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(linearQ_count_cuda, linearQ_count, totalQuery*sizeof(int), hipMemcpyHostToDevice);

	gettimeofday(&transferend, NULL);

	gettimeofday(&gpustart, NULL);
	dim3  block(THREADS_PER_BLOCK, 1);
	dim3  grid((totalQuery + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK, 1);
      pirntf("Query Number: %d - Block Number %d\n", totalQuery, grid.x);
      
	SvS_GPU<<<grid, block>>>(
		queryLength_counter,
		queryLength->vocabSize,//queryLength_vocabSize,
		queryLength->defaultValue,//queryLength_defaultValue,	
		idToIndexMap_counter,
		idToIndexMap->vocabSize,
		idToIndexMap->defaultValue,		
		index_df_counter,
		index->pointers->df->vocabSize,//_df_vocabSize,
		index->pointers->df->defaultValue,		
		index_pointer_counter,
		index->pointers->startPointers->vocabSize,
		index->pointers->startPointers->defaultValue,
		index_pool_firstseg, //index->pool->pool[0]
		index->pool->offset,
		index->pool->segment,
		linearQ_cuda,
		linearQ_count_cuda,
		totalQuery);

	gettimeofday(&gpuend, NULL);

	printf("Transfer Timing: %10.0f\n",
		   ((float) ((transferend.tv_sec * 1000000 + transferend.tv_usec) -
					 (transferstart.tv_sec * 1000000 + transferstart.tv_usec))));
	printf("GPU Timing: %10.0f\n",
		   ((float) ((gpuend.tv_sec * 1000000 + gpuend.tv_usec) -
					 (gpustart.tv_sec * 1000000 + gpustart.tv_usec))));

}



int main (int argc, char** args) {
  // Index path
  char* inputPath = getValueCL(argc, args, "-index");
  // Query path
  char* queryPath = getValueCL(argc, args, "-query");
  // Output path (optional)
  char* outputPath = getValueCL(argc, args, "-output");
  // Hits
  int hits = 1000;
  if(isPresentCL(argc, args, "-hits")) {
    hits = atoi(getValueCL(argc, args, "-hits"));
  }
  // Algorithm
  //char* intersectionAlgorithm = getValueCL(argc, args, "-algorithm");
  //Algorithm algorithm = SVS;

  // Algorithm is limited to the following list (case sensitive):
  // - SvS (conjunctive)
  // - WAND (disjunctive)
  /*if(!strcmp(intersectionAlgorithm, "SvS")) {
    algorithm = SVS;
  } else if(!strcmp(intersectionAlgorithm, "WAND")) {
    algorithm = WAND;
  } else {
    printf("Invalid algorithm (Options: SvS | WAND)\n");
    return;
  }*/

  // Read the inverted index
  printf("Start reading!\n");
  InvertedIndex* index = readInvertedIndex(inputPath);
  printf("Done reading!\n");
  // Read queries. Query file must be in the following format:
  // - First line: <number of queries: integer>
  // - <query id: integer> <query length: integer> <query text: string>
  // Note that, if a query term does not have a corresponding postings list,
  // then we drop the query term from the query. Empty queries are not evaluated.
  FixedIntCounter* queryLength = createFixedIntCounter(32768, ZERO);
  FixedIntCounter* idToIndexMap = createFixedIntCounter(32768, ZERO);
  FILE* fp = fopen(queryPath, "r");
  int totalQueries = 0, id, qlen, fqlen, j, pos, termid, i;
  char query[1024];
  fscanf(fp, "%d", &totalQueries);
  //unsigned int** queries = (unsigned int**) malloc(totalQueries * sizeof(unsigned int*));
  unsigned int* linearQ = (unsigned int*) malloc(100 * totalQueries * sizeof(unsigned int));
  int* linearQ_count = (int*) malloc(totalQueries * sizeof(unsigned int));

  int totalLen = 0;
  for(i = 0; i < totalQueries; i++) {
    fscanf(fp, "%d %d", &id, &qlen);
    //queries[i] = (unsigned int*) malloc(qlen * sizeof(unsigned int));
    pos = 0;
    fqlen = qlen;
    for(j = 0; j < qlen; j++) {
      fscanf(fp, "%s", query);
      termid = getTermId(index->dictionary, query);
      if(termid >= 0) {
        if(getStartPointer(index->pointers, termid) != UNDEFINED_POINTER) {
			linearQ[totalLen] = termid;
			totalLen++;
          //queries[i][pos++] = termid;
        } else {
          fqlen--;
        }
      } else {
        fqlen--;
      }
    }
    setFixedIntCounter(idToIndexMap, id, i);
    setFixedIntCounter(queryLength, id, fqlen);
	linearQ_count[i] = totalLen;
  }
  fclose(fp);

  if(outputPath) {
    fp = fopen(outputPath, "w");
  }

  // Evaluate queries by iterating over the queries that are not empty
  id = -1;

/////////////////////// CUDA Entry
  SvS_GPU_Entry(
  	queryLength, 
  	idToIndexMap, 
  	outputPath, 
  	index, 
  	fp, 
  	totalQueries,
  	linearQ,
  	linearQ_count,
  	totalLen);
//////////////////////

  if(outputPath) {
    fclose(fp);
  }
  /*for(i = 0; i < totalQueries; i++) {
    if(queries[i]) {
      free(queries[i]);
    }
  }
  free(queries);*/
  destroyFixedIntCounter(queryLength);
  destroyFixedIntCounter(idToIndexMap);
  destroyInvertedIndex(index);
  return 0;
}


